
#include <hip/hip_runtime.h>
/*++

Copyright (c) 2017 Trent Nelson <trent@trent.me>

Module Name:

    TraceStoreKernels.cu

Abstract:

    This module implements CUDA kernels for various trace store functions.

--*/

#ifdef __cplusplus
extern "C" {
#endif

__global__
void saxpy(int n, float a, float *x, float *y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n;
         i += blockDim.x * gridDim.x) {

        y[i] = a * x[i] + y[i];
    }
}

#ifdef __cplusplus
}
#endif

// vim:set ts=8 sw=4 sts=4 tw=80 expandtab                                     :
